#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdio>
#include <hipblas.h>

#include "tensor.h"

void _vector_add(int n, float *a, float *b, float *y) {
  for (int i = 0; i < n; i++) {
    y[i] = a[i] + b[i];
  }
}

void vector_add(VectorAddOutputMode output_mode) {
  int n = 10'000'000;

  float *a, *b, *y;
  a = (float *)malloc(sizeof(float) * n);
  b = (float *)malloc(sizeof(float) * n);
  y = (float *)malloc(sizeof(float) * n);

  for (int i = 0; i < n; i++) {
    a[i] = 1.f;
    b[i] = 2.f;
  }

  _vector_add(n, a, b, y);

  if (output_mode == PRINT) {
    printf("%f, %f, %f\n", y[0], y[n / 2], y[n - 1]);
  } else if (output_mode == TEST) {
    for (int i = 0; i < n; i++) {
      assert(fabs(y[i] - a[i] - b[i]) < 1e-6);
    }

    printf("All values are correct.\n");
  }

  free(a);
  free(b);
  free(y);
}

__global__ void _cuda_vector_add(int n, float *a, float *b, float *y) {
  for (int i = 0; i < n; i++) {
    y[i] = a[i] + b[i];
  }
}

__global__ void _cuda_vector_add_sb(int n, float *a, float *b, float *y) {
  // 0, 256, 512, 768, 1024, ...
  int index = threadIdx.x;
  int stride = blockDim.x;

  for (int i = index; i < n; i += stride) {
    y[i] = a[i] + b[i];
  }
}

__global__ void _cuda_vector_add_mb(int n, float *a, float *b, float *y) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = index; i < n; i += stride) {
    y[i] = a[i] + b[i];
  }
}

void cuda_vector_add(VectorAddVariant variant, VectorAddOutputMode output_mode) {
  int n = 10'000'000;

  float *a, *b, *y;
  a = (float *)malloc(sizeof(float) * n);
  b = (float *)malloc(sizeof(float) * n);
  y = (float *)malloc(sizeof(float) * n);

  for (int i = 0; i < n; i++) {
    a[i] = 1.f;
    b[i] = 2.f;
  }

  float *device_a, *device_b, *device_y;
  hipMalloc((void **)&device_a, sizeof(float) * n);
  hipMemcpy(device_a, a, sizeof(float) * n, hipMemcpyHostToDevice);

  hipMalloc((void **)&device_b, sizeof(float) * n);
  hipMemcpy(device_b, b, sizeof(float) * n, hipMemcpyHostToDevice);

  hipMalloc((void **)&device_y, sizeof(float) * n);

  if (variant == SINGLE_THREAD) {
    _cuda_vector_add<<<1, 1>>>(n, device_a, device_b, device_y);
  } else if (variant == SINGLE_BLOCK) {
    _cuda_vector_add_sb<<<1, 256>>>(n, device_a, device_b, device_y);
  } else if (variant == MULTIPLE_BLOCKS) {
    int n_blocks = (n + 256 - 1) / 256;
    _cuda_vector_add_mb<<<n_blocks, 256>>>(n, device_a, device_b, device_y);
  }

  hipDeviceSynchronize();

  if (output_mode == PRINT) {
    float out_a, out_b, out_c;
    hipMemcpy(&out_a, device_y, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&out_b, device_y + (n / 2), sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&out_c, device_y + (n - 1), sizeof(float), hipMemcpyDeviceToHost);

    printf("%f, %f, %f\n", out_a, out_b, out_c);
  } else if (output_mode == TEST) {
    hipMemcpy(y, device_y, sizeof(float) * n, hipMemcpyDeviceToHost);
    for (int i = 0; i < n; i++) {
      assert(fabs(y[i] - a[i] - b[i]) < 1e-6);
    }

    printf("All values are correct.\n");
  }

  hipFree(device_a);
  free(a);

  hipFree(device_b);
  free(b);

  hipFree(device_y);
  free(y);
}

void _vector_dot(int n, float *a, float *b, float *y) {
  for (int i = 0; i < n; i++) {
    *y += a[i] * b[i];
  }
}

void vector_dot() {
  int n = 10'000'000;

  float *a, *b, *y;
  a = (float *)malloc(sizeof(float) * n);
  b = (float *)malloc(sizeof(float) * n);
  y = (float *)malloc(sizeof(float));

  *y = 0.f;
  for (int i = 0; i < n; i++) {
    a[i] = 1.f;
    b[i] = 2.f;
  }

  _vector_dot(n, a, b, y);
  printf("%f\n", *y);

  free(a);
  free(b);
  free(y);
}

__global__ void _cuda_vector_dot(int n, float *a, float *b, float *y) {
  __shared__ float cache[256];

  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  float temp = 0.f;
  for (int i = index; i < n; i += stride) {
    temp += a[i] * b[i];
  }

  cache[threadIdx.x] = temp;
  __syncthreads();

  for (int i = blockDim.x / 2; i > 0; i >>= 1) {
    if (threadIdx.x < i)
      cache[threadIdx.x] += cache[threadIdx.x + i];
    __syncthreads();
  }

  if (threadIdx.x == 0)
    y[blockIdx.x] = cache[0];
}

void cuda_vector_dot() {
  int n = 10'000'000;
  int n_blocks = (n + 256 - 1) / 256;

  float *a, *b, *y;
  hipMallocManaged(&a, sizeof(float) * n);
  hipMallocManaged(&b, sizeof(float) * n);
  hipMallocManaged(&y, sizeof(float) * n_blocks);

  for (int i = 0; i < n; i++) {
    a[i] = 1.f;
    b[i] = 2.f;
  }

  _cuda_vector_dot<<<n_blocks, 256>>>(n, a, b, y);
  hipDeviceSynchronize();

  float c = 0.f;
  for (int i = 0; i < n_blocks; i++)
    c += y[i];

  printf("%f\n", c);

  hipFree(a);
  hipFree(b);
  hipFree(y);
}

void cublas_vector_dot() {
  int n = 10'000'000;

  float *a, *b, *y;
  hipMallocManaged(&a, sizeof(float) * n);
  hipMallocManaged(&b, sizeof(float) * n);
  hipMallocManaged(&y, sizeof(float));

  *y = 0.f;
  for (int i = 0; i < n; i++) {
    a[i] = 1.f;
    b[i] = 2.f;
  }

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSdot(handle, n, a, 1, b, 1, y);
  hipblasDestroy(handle);

  printf("%f\n", *y);

  hipFree(a);
  hipFree(b);
  hipFree(y);
}
